#include "hip/hip_runtime.h"
#include <pycuda-complex.hpp>
#include <math.h>
#include <hip/hip_complex.h>
typedef pycuda::complex<double> cmplx;

/*
    compute P_eph^{(1)} -> e-ph scattering first order
*/

__global__ void compute_P1_eph(int nm, int nst, cmplx *P) {
    /*internal variables */
    const int i = threadIdx.x + blockDim.x * blockIdx.x;
    const int j = threadIdx.y + blockDim.y * blockIdx.y;
    const int k = threadIdx.z + blockDim.z * blockIdx.z;
    const int idx = i + j * blockDim.x * gridDim.x + k * blockDim.x * gridDim.x * blockDim.y * gridDim.y;
    if (idx == 0) {
        printf("%d    -     %d\n", nm, nst);
    } 
}