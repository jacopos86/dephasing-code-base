#include "hip/hip_runtime.h"
#include <pycuda-complex.hpp>
#include <math.h>
#include "extern_func.cuh"
#define PI 3.141592653589793
typedef pycuda::complex<double> cmplx;

__global__ void compute_acf_V2_oft(int *qlp_init, int *lgth, int *qlp_lst, const int SIZE,
double *time, double wq, double *wqp, double wuq, double *wuqp, double Alq, double *Alqp, 
cmplx *Flqlqp, double T, double DE, double NU, double MINFREQ, double THZTOEV, double KB, 
const double TOLER, cmplx *acf, cmplx *acf_int) {
    const int i = threadIdx.x + blockDim.x * blockIdx.x;
    const int j = threadIdx.y + blockDim.y * blockIdx.y;
    const int k = threadIdx.z + blockDim.z * blockIdx.z;
    //printf("%d ", blockIdx.z);
    int idx = i + j * blockDim.x * gridDim.x + k * blockDim.x * gridDim.x * blockDim.y * gridDim.y;
    int tx  = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;
    int iqlx= blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.x * gridDim.y;
    /* internal variables */

    if (tx < SIZE) {
        iqlp0 = qlp_init[iqlx];
        n = lgth[iqlx];
        wql = 2.*PI*wuq;
        /* ph. occup. */
        Eql = wuq * THZTOEV;
        x = Eql / (KB * T);
        nql = bose_occup(x, T, TOLER);
        /* run over (q',l') */
        for (ii=iqlp0; ii<iqlp0+n; ii++) {
            iqlp = qlp_lst[ii]; 
            if (wuqp[iqlp] > MINFREQ) {
                wqlp = 2.*PI*wuqp[iqlp];
                Eqlp = wuqp[iqlp] * THZTOEV;
                /* set e^iwt */
                re = cos((DE + wqlp - wql)*time[tx]);
                im = sin((DE + wqlp - wql)*time[tx]);
                cmplx eiwt(re, -im);
                /* ph. occup.*/
                x = Eqlp / (KB * T);
                nqlp = bose_occup(x, T, TOLER);
                /* acf^2(t) */
                ft = nql * (1.+nqlp) * eiwt * EXP(-NU*time[tx]);
                acf[idx] += wq * wqp[iqlp] * Alq * Alq * Alqp[iqlp] * Alqp[iqlp] * ft * Flqlqp[iqlp] * conj(Flqlqp[iqlp]);
                /* \int acf^2(t) */
                ft(0.,0.);
                cmplx DN(DE+wqlp-wql, -NU); 
                ft = IU * nql * (1.+nqlp) * (eiwt * EXP(-NU*time[tx]) - 1.) / DN;
                acf_int[idx] += wq * wqp[iqlp] * Alq * Alq * Alqp[iqlp] * Alqp[iqlp] * ft * Flqlqp[iqlp] * conj(Flqlqp[iqlp]);
            }
        }
    }
}

__global__ void compute_acf_V2_ofw(int *qlp_init, int *lgth, int *qlp_lst, const int SIZE,
double *wg, double wq, double *wqp, double wuq, double *wuqp, double Alq, double *Alqp,
cmplx *Flqlqp, double T, double DE, double MINFREQ, double THZTOEV, double KB, 
const double TOLER, double ETA, cmplx *acfw) {
    const int i = threadIdx.x + blockDim.x * blockIdx.x;
    const int j = threadIdx.y + blockDim.y * blockIdx.y;
    const int k = threadIdx.z + blockDim.z * blockIdx.z;
    int idx = i + j * blockDim.x * gridDim.x + k * blockDim.x * gridDim.x * blockDim.y * gridDim.y;
    int iwx = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;
    int iqlpx= blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.x * gridDim.y;
    /* internal variables */


    /* check iwx */
    if (iwx < SIZE) {
        iqlp0 = qlp_init[iqlpx];
        n = lgth[iqlpx];
        wql = 2.*PI*wuq;
        /* ph. occup.*/
        Eql = wuq * THZTOEV;
        x = Eql / (KB * T);
        nql = bose_occup(x, T, TOLER);
        /* cycle (q',l') */
        for (ii=iqlp0; ii<iqlp0+n; ii++) {
            iqlp = qlp_lst[ii];
            if (wuqp[iqlp] > MINFREQ) {
                wqlp = 2.*PI*wuqp[iqlp];
                Eqlp = wuqp[iqlp] * THZTOEV;
                /* ph. occup. */
                x = Eqlp / (KB * T);
                nqlp = bose_occup(x, T, TOLER);
                /* compute lorentzian */
                x = DE + Eqlp - Eql + wg[iwx];
                LTZ = lorentzian(x, ETA);
                fw = nql * (1. + nqlp) * LTZ;
                /* compute ACF */
                acfw[idx] += wq * wqp[iqlp] * Alq * Alq * Alqp[iqlp] * Alqp[iqlp] * fw * Flqlqp[iqlp] * conj(Flqlqp[iqlp]);
            }
        }
    }
}

__global__ void compute_acf_V2_phr_oft(int *qlp_init, int *lgth, int *qlp_lst, double *time, const int SIZE,
double DE, double NU, double wq, double wuq, double Alq, double *wqp, double *wuqp, double *Alqp, cmplx *Flqlqp,
double T, double MINFREQ, double THZTOEV, double KB, const double TOLER, cmplx *acf, cmplx *acf_int) {
    const int i = threadIdx.x + blockDim.x * blockIdx.x;
    const int j = threadIdx.y + blockDim.y * blockIdx.y;
    const int k = threadIdx.z + blockDim.z * blockIdx.z;
    int idx = i + j * blockDim.x * gridDim.x + k * blockDim.x * gridDim.x * blockDim.y * gridDim.y;
    int tx  = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;
    int iqlx= blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.x * gridDim.y;
    /* internal variables */




    if (tx < SIZE) {
        
    }



}