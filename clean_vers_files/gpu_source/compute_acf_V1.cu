#include "hip/hip_runtime.h"
#include <pycuda-complex.hpp>
#include <math.h>
#include "LOCAL-PATH/extern_func.cuh"
#define PI 3.141592653589793
typedef pycuda::complex<double> cmplx;

/* time ACF first order */

__global__ void compute_acf_V1_oft(int *ql_init, int *lgth, int *ql_lst, const int SIZE,
double *time, double *wq, double *wuq, double *Alq, cmplx *Flq, double T, double DE, double NU,
double MINFREQ, double THZTOEV, double KB, const double TOLER, cmplx *acf, cmplx *acf_int) {
    const int i = threadIdx.x + blockDim.x * blockIdx.x;
    const int j = threadIdx.y + blockDim.y * blockIdx.y;
    const int k = threadIdx.z + blockDim.z * blockIdx.z;
    int idx = i + j * blockDim.x * gridDim.x + k * blockDim.x * gridDim.x * blockDim.y * gridDim.y;
    int tx  = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;
    int iqlx= blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.x * gridDim.y;
    /* internal variables */
    double x, wql, Eql, nql;
    int iql0, iql, n, ii;
    double re, im;
    cmplx ft;
    const cmplx IU(0., 1.);
    /* run over t and iql index */
    if (tx < SIZE) {
        iql0= ql_init[iqlx];
        n = lgth[iqlx];
        for (ii=iql0; ii<iql0+n; ii++) {
            iql = ql_lst[ii];
            if (wuq[iql] > MINFREQ) {
                wql= 2.*PI*wuq[iql]; 
                re = cos((wql+DE)*time[tx]);
                im = sin((wql+DE)*time[tx]);
                cmplx eiwt(re, -im);
                re = cos((wql-DE)*time[tx]);
                im = sin((wql-DE)*time[tx]);
                cmplx cc_eiwt(re, im);
                /* ph. occup */
                Eql = wuq[iql] * THZTOEV;
                x = Eql / (KB * T);
                nql = bose_occup(x, T, TOLER);
                /* compute auto correl functions (eV^2) units */
                ft = ((1. + nql) * eiwt + nql * cc_eiwt) * exp(-NU*time[tx]);
                acf[idx] += wq[iql] * Alq[iql] * Alq[iql] * ft * Flq[iql] * conj(Flq[iql]);
                /* compute cumulative sum auto correl function (eV^2 ps) units */
                cmplx DN1(wql+DE,-NU);
                cmplx DN2(wql-DE, NU);
                ft = IU * (1. + nql) * (eiwt * exp(-NU*time[tx]) - 1.) / DN1 - IU * nql * (cc_eiwt * exp(-NU*time[tx]) - 1.) / DN2;
                acf_int[idx] += wq[iql] * Alq[iql] * Alq[iql] * ft * Flq[iql] * conj(Flq[iql]);
            }
        }
    }
}

/* freq. ACF order 1 */
__global__ void compute_acf_V1_ofw(int *ql_init, int *lgth, int *ql_lst, const int SIZE, double *wg,
double *wq, double *wuq, double *Alq, cmplx *Flq, double T, double DE, 
double MINFREQ, double THZTOEV, double KB, const double TOLER, double ETA, cmplx *acfw) {
    const int i = threadIdx.x + blockDim.x * blockIdx.x;
    const int j = threadIdx.y + blockDim.y * blockIdx.y;
    const int k = threadIdx.z + blockDim.z * blockIdx.z;
    int idx = i + j * blockDim.x * gridDim.x + k * blockDim.x * gridDim.x * blockDim.y * gridDim.y;
    int iwx = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;
    int iqlx= blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.x * gridDim.y;
    /* internal variables */
    int iql0, iql, ii, n;
    double x, Eql, nql;
    double LTZ1, LTZ2, fw;
    /* run over w - ql index */
    if (iwx < SIZE) {
        iql0 = ql_init[iqlx];
        n = lgth[iqlx];
        for (ii=iql0; ii<iql0+n; ii++) {
            iql = ql_lst[ii];
            if (wuq[iql] > MINFREQ) {
                /* Eql in eV */
                Eql= wuq[iql] * THZTOEV;
                x = Eql / (KB * T);
                nql = bose_occup(x, T, TOLER);
                /* compute lorentzians */
                x = DE + Eql + wg[iwx];
                LTZ1 = lorentzian(x, ETA);
                x = DE - Eql + wg[iwx];
                LTZ2 = lorentzian(x, ETA);
                fw = (1.+nql) * LTZ1 + nql * LTZ2;
                /* compute ACF */
                acfw[idx] += wq[iql] * Alq[iql] * Alq[iql] * fw * Flq[iql] * conj(Flq[iql]);
            }
        }
    }
}

/* atom res. ACF of t*/

__global__ void compute_acf_V1_atr_oft(int *at_lst, double *wq, double *wuq, double *time,
double DE, double NU, cmplx *Fjax_lq, double *Alq, int SIZE, int NA_SIZE, int NMODES, int NAT, 
double T, double MINFREQ, double THZTOEV, double KB, double TOLER, cmplx *acf, cmplx *acf_int) {
    const int i = threadIdx.x + blockDim.x * blockIdx.x;
    const int j = threadIdx.y + blockDim.y * blockIdx.y;
    const int k = threadIdx.z + blockDim.z * blockIdx.z;
    int idx = i + j * blockDim.x * gridDim.x + k * blockDim.x * gridDim.x * blockDim.y * gridDim.y;
    int tx = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;
    int ax = blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.x * gridDim.y;
    /* internal vars*/
    int iql;
    int dx, iFx;
    double re, im;
    double wql, Eql, x, nql;
    cmplx ft;
    const cmplx IU(0., 1.);
    /* check tx < SIZE */
    if (tx < SIZE && ax < NA_SIZE) {
        int ia = at_lst[ax];
        /* iterate over (q,l) */
        for (iql=0; iql<NMODES; iql++) {
            if (wuq[iql] > MINFREQ) {
                wql = 2.*PI*wuq[iql];
                re = cos((wql+DE)*time[tx]);
                im = sin((wql+DE)*time[tx]);
                cmplx eiwt(re, -im);
                re = cos((wql-DE)*time[tx]);
                im = sin((wql-DE)*time[tx]);
                cmplx cc_eiwt(re, im);
                /* ph. occup. */
                Eql = wuq[iql] * THZTOEV;
                x = Eql / (KB * T);
                nql = bose_occup(x, T, TOLER);
                /* compute auto correl functions (eV^2) units */
                ft = ((1. + nql) * eiwt + nql * cc_eiwt) * exp(-NU*time[tx]);
                for (dx=0; dx<3; dx++) {
                    iFx = 3*NAT*iql+3*ia+dx;
                    acf[idx] += wq[iql] * Alq[iql] * Alq[iql] * ft * Fjax_lq[iFx] * conj(Fjax_lq[iFx]);
                }
                /* compute cumulative sum auto correl function (eV^2 ps) units */
                cmplx DN1(wql+DE,-NU);
                cmplx DN2(wql-DE, NU);
                ft = IU * (1. + nql) * (eiwt * exp(-NU*time[tx]) - 1.) / DN1 - IU * nql * (cc_eiwt * exp(-NU*time[tx]) - 1.) / DN2;
                for (dx=0; dx<3; dx++) {
                    iFx = 3*NAT*iql+3*ia+dx;
                    acf_int[idx] += wq[iql] * Alq[iql] * Alq[iql] * ft * Fjax_lq[iFx] * conj(Fjax_lq[iFx]);
                }
            }
        }
    }
}

/* atom res. ACF of t*/

__global__ void compute_acf_V1_phr_oft(int NPH, int *ph_lst, int SIZE, double *time, double *wq,
double *wuq, double *Alq, cmplx *Flq, double T, double DE, double NU, double MINFREQ, double THZTOEV, 
double KB, double TOLER, cmplx *acf, cmplx *acf_int) {
    const int i = threadIdx.x + blockDim.x * blockIdx.x;
    const int j = threadIdx.y + blockDim.y * blockIdx.y;
    const int k = threadIdx.z + blockDim.z * blockIdx.z;
    int idx= i + j * blockDim.x * gridDim.x + k * blockDim.x * gridDim.x * blockDim.y * gridDim.y;
    int tx = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;
    int phx= blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.x * gridDim.y;
    int iql = ph_lst[phx];
    /* internal vars*/
    double wql, Eql, x, nql;
    double re, im;
    cmplx ft;
    const cmplx IU(0., 1.);
    /* check tx < SIZE */
    if (tx < SIZE && phx < NPH) {
        if (wuq[iql] > MINFREQ) {
            wql = 2.*PI*wuq[iql];
            re = cos((wql+DE)*time[tx]);
            im = sin((wql+DE)*time[tx]);
            cmplx eiwt(re, -im);
            re = cos((wql-DE)*time[tx]);
            im = sin((wql-DE)*time[tx]);
            cmplx cc_eiwt(re, im);
            /* ph. occup.*/
            Eql = wuq[iql] * THZTOEV;
            x = Eql / (KB * T);
            nql = bose_occup(x, T, TOLER);
            /* compute ACF (eV^2) units*/
            ft = ((1. + nql) * eiwt + nql * cc_eiwt) * exp(-NU*time[tx]);
            acf[idx] += wq[iql] * Alq[iql] * Alq[iql] * ft * Flq[iql] * conj(Flq[iql]);
            /* compute cumulative sum auto correl function (eV^2 ps) units */
            cmplx DN1(wql+DE,-NU);
            cmplx DN2(wql-DE, NU);
            ft = IU * (1. + nql) * (eiwt * exp(-NU*time[tx]) - 1.) / DN1 - IU * nql * (cc_eiwt * exp(-NU*time[tx]) - 1.) / DN2;
            acf_int[idx] += wq[iql] * Alq[iql] * Alq[iql] * ft * Flq[iql] * conj(Flq[iql]);
        }
    }
}

/* freq. ACF order 1 -> ATR */
__global__ void compute_acf_V1_atr_ofw(int NA_SIZE, int *at_lst, const int SIZE, double *wg, double *wq, 
double *wuq, double *Alq, cmplx *Fjax_lq, double T, double DE, int NMODES, int NAT,
double MINFREQ, double THZTOEV, double KB, const double TOLER, double ETA, cmplx *acfw) {
    const int i = threadIdx.x + blockDim.x * blockIdx.x;
    const int j = threadIdx.y + blockDim.y * blockIdx.y;
    const int k = threadIdx.z + blockDim.z * blockIdx.z;
    int idx = i + j * blockDim.x * gridDim.x + k * blockDim.x * gridDim.x * blockDim.y * gridDim.y;
    int iwx = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;
    int ax = blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.x * gridDim.y;
    int ia = at_lst[ax];
    /* internal variables */
    int iql;
    int dx, iFx;
    double Eql, x, nql;
    double LTZ1, LTZ2, fw;
    /* tx - na SIZE*/
    if (iwx < SIZE && ax < NA_SIZE) {
        for (iql=0; iql<NMODES; iql++) {
            if (wuq[iql] > MINFREQ) {
                /* Eql in eV*/
                Eql = wuq[iql] * THZTOEV;
                x = Eql / (KB * T);
                nql = bose_occup(x, T, TOLER);
                /* compute lorentzians */
                x = DE + Eql + wg[iwx];
                LTZ1 = lorentzian(x, ETA);
                x = DE - Eql + wg[iwx];
                LTZ2 = lorentzian(x, ETA);
                fw = (1.+nql) * LTZ1 + nql * LTZ2;
                /* compute ACF(w) - eV*/
                for (dx=0; dx<3; dx++) {
                    iFx = 3*NAT*iql+3*ia+dx;
                    acfw[idx] += wq[iql] * Alq[iql] * Alq[iql] * fw * Fjax_lq[iFx] * conj(Fjax_lq[iFx]);
                }
            }
        }
    }
}

/* freq. ACF order 1 -> PHR */
__global__ void compute_acf_V1_phr_ofw(int NPH, int *ph_lst, const int SIZE, double *wg, double *wq,
double *wuq, double *Alq, cmplx *Flq, double T, double DE, double MINFREQ, double THZTOEV, 
double KB, const double TOLER, double ETA, cmplx *acfw) {
    const int i = threadIdx.x + blockDim.x * blockIdx.x;
    const int j = threadIdx.y + blockDim.y * blockIdx.y;
    const int k = threadIdx.z + blockDim.z * blockIdx.z;
    int idx = i + j * blockDim.x * gridDim.x + k * blockDim.x * gridDim.x * blockDim.y * gridDim.y;
    int iwx = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;
    int phx = blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.x * gridDim.y;
    int iql = ph_lst[phx];
    /* internal variables*/
    double Eql, x, nql;
    double LTZ1, LTZ2, fw;
    /* check tx size*/
    if (iwx < SIZE && phx < NPH) {
        if (wuq[iql] > MINFREQ) {
            /* Eql in eV*/
            Eql = wuq[iql] * THZTOEV;
            x = Eql / (KB * T);
            nql = bose_occup(x, T, TOLER);
            /* compute lorentzians*/
            x = DE + Eql + wg[iwx];
            LTZ1 = lorentzian(x, ETA);
            x = DE - Eql + wg[iwx];
            LTZ2 = lorentzian(x, ETA);
            /* compute ACF(w) - eV*/
            fw = (1.+nql) * LTZ1 + nql * LTZ2;
            acfw[idx] += wq[iql] * Alq[iql] * Alq[iql] * fw * Flq[iql] * conj(Flq[iql]);
        }
    }
}